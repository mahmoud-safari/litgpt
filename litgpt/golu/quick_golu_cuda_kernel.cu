#include "hip/hip_runtime.h"
// References - https://github.com/pytorch/pytorch
// Check https://github.com/pytorch/pytorch/tree/main/aten/src/ATen/native/cuda for more details

#define TORCH_ASSERT_NO_OPERATORS
#define _USE_MATH_DEFINES

#include <cmath>

#include <thrust/tuple.h>

#include <ATen/native/Activation.h>
#include <ATen/AccumulateType.h>
#include <ATen/Dispatch.h>
#include <ATen/core/TensorBase.h>
#include <ATen/cuda/ApplyGridUtils.cuh>
#include <ATen/cuda/detail/OffsetCalculator.cuh>
#include <ATen/native/cuda/Loops.cuh>

#include <c10/core/Scalar.h>
#include <c10/cuda/CUDAMathCompat.h>


extern "C" {

// GoLU Backward CUDA Kernel Implementation
void QuickGoLUBackwardCUDAKernelImpl(at::TensorIteratorBase& iter, double alpha, double beta, double gamma) {
    AT_DISPATCH_FLOATING_TYPES_AND2(
        at::ScalarType::Half, at::ScalarType::BFloat16, iter.dtype(), "quick_golu_backward_cuda", [&]() {
        
        // This is a type alias which allows you to define a new name (opmath_t) for an existing type
        using opmath_t = at::opmath_type<scalar_t>;
        
        // Have these as globals to avoid multiple lambda reallocation
        const opmath_t alpha_ = static_cast<opmath_t>(alpha);
        const opmath_t beta_ = static_cast<opmath_t>(beta);
        const opmath_t gamma_ = static_cast<opmath_t>(gamma);
        
        at::native::gpu_kernel(iter, [alpha_, beta_, gamma_] GPU_LAMBDA(scalar_t dy, scalar_t x) -> scalar_t {
            
            // Cast the input variables to correct datatype
            // This is an inplace operation to reduce memory footprint
            dy = static_cast<opmath_t>(dy);
            x = static_cast<opmath_t>(x);
            
            // Compute common terms in grad_x to reuse them - reduces 3 exp calculations to 2
            opmath_t inner_exp = c10::cuda::compat::exp(-gamma_ * x);
            
            // Compute the respective gradients for x
            opmath_t grad_x = dy * alpha_ * c10::cuda::compat::exp(-beta_ * inner_exp) * (
                opmath_t(1) + beta_ * gamma_ * x * inner_exp);
            
            // NaN is the only value in IEEE Floating point which is not equal to itself
            grad_x = (grad_x != grad_x) ? opmath_t(0) : grad_x;

            return static_cast<opmath_t>(grad_x);
        });
    });
}


} // extern "C"
